#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>
#include <boost/tokenizer.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <string>
#include <cstring>
#include <cctype>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <bitset>
#include <random>
#include "timer.h"

using namespace std;

__device__ float
squared_l2_distance(float x_1, float y_1, float x_2, float y_2) {
  return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

float
squared_l2_distance_h(float x_1, float y_1, float x_2, float y_2) {
  return (x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2);
}

__global__ void assign_clusters(const thrust::device_ptr<float> data_x,
                                const thrust::device_ptr<float> data_y,
                                int data_size,
                                const thrust::device_ptr<float> means_x,
                                const thrust::device_ptr<float> means_y,
                                thrust::device_ptr<float> new_sums_x,
                                thrust::device_ptr<float> new_sums_y,
                                int k,
                                thrust::device_ptr<int> counts,
				thrust::device_ptr<int> d_clusterNo) {
				
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= data_size) return;

  // Make global loads once.
  const float x = data_x[index];
  const float y = data_y[index];

  float best_distance = FLT_MAX;
  int best_cluster = 0;
  for (int cluster = 0; cluster < k; ++cluster) {
    const float distance =
        squared_l2_distance(x, y, means_x[cluster], means_y[cluster]);
    if (distance < best_distance) {
      best_distance = distance;
      best_cluster = cluster;
    }
  }

  // d_clusterNo[index] = best_cluster;
  //  __syncthreads();

  atomicAdd(thrust::raw_pointer_cast(new_sums_x + best_cluster), x);
  atomicAdd(thrust::raw_pointer_cast(new_sums_y + best_cluster), y);
  atomicAdd(thrust::raw_pointer_cast(counts + best_cluster), 1);
}

__global__ void compute_new_means(thrust::device_ptr<float> means_x,
                                  thrust::device_ptr<float> means_y,
                                  const thrust::device_ptr<float> new_sum_x,
                                  const thrust::device_ptr<float> new_sum_y,
                                  const thrust::device_ptr<int> counts) {
  const int cluster = threadIdx.x;
  const int count = max(1, counts[cluster]);
  means_x[cluster] = new_sum_x[cluster] / count;
  means_y[cluster] = new_sum_y[cluster] / count;
}

void sort(unsigned long long *key, long *value, unsigned long long *key_out, long *value_out, int kBytes, int vBytes, size_t data_size, int thread_id)
{
    int GPU_number = thread_id % 4;

    // cout << "transfer:threadID:" << thread_id << ",data size:" << "," << data_size << endl;

    thrust::host_vector<unsigned long long> h_vec_key(data_size);
    thrust::host_vector<long> h_vec_value(data_size);

    for(int i=0; i < data_size; i++)
    {
	h_vec_key[i] = key[i];
	h_vec_value[i] = value[i];
    }

    hipSetDevice(GPU_number);
    
    thrust::device_vector<unsigned long long> d_vec_key(data_size);
    thrust::device_vector<long> d_vec_value(data_size);

    thrust::copy(h_vec_key.begin(), h_vec_key.end(), d_vec_key.begin());
    thrust::copy(h_vec_value.begin(), h_vec_value.end(), d_vec_value.begin());
    
    thrust::sort_by_key(d_vec_key.begin(), d_vec_key.end(), d_vec_value.begin());

    thrust::host_vector<unsigned long long> h_vec_key_2(data_size);
    thrust::host_vector<long> h_vec_value_2(data_size);

    thrust::copy(d_vec_value.begin(),d_vec_value.end(),h_vec_value_2.begin());
    thrust::copy(d_vec_key.begin(),d_vec_key.end(),h_vec_key_2.begin());

    /*
    for(int i = 0; i < 3; i++)
    {
	cout << "[sort result] threadID:" << thread_id << ":" << h_vec_key_2[i] << ","
	     << h_vec_value_2[i] << endl;
    }
    */
    
    for(int i = 0; i < data_size; i++)
    {
    	key_out[i] =  h_vec_key_2[i];
	value_out[i] =  h_vec_value_2[i];
    }
}

void transfer(unsigned long long *key_1, float *value_1, unsigned long long *key_2, float *value_2, unsigned long long *key_out, float *value_out, int kBytes, int vBytes, size_t data_size, int *new_size, int thread_id)
{
    // unsigned int t, travdirtime;
    int GPU_number = thread_id % 4;

    clock_t start_t = clock();

    thrust::host_vector<unsigned long long> h_vec_key_1(data_size);
    thrust::host_vector<float> h_vec_value_1(data_size);
    for(int i=0; i < data_size; i++)
    {
	h_vec_key_1[i] = key_1[i];
	h_vec_value_1[i] = value_1[i];
    }

    thrust::host_vector<unsigned long long> h_vec_key_2(data_size);
    thrust::host_vector<float> h_vec_value_2(data_size);
    for(int i=0; i < data_size; i++)
    {
	h_vec_key_2[i] = key_2[i];
	h_vec_value_2[i] = value_2[i];
    }

    //start_timer(&t);
    hipSetDevice(GPU_number);
    
    thrust::device_vector<unsigned long long> d_vec_key_1(data_size);
    thrust::device_vector<float> d_vec_value_1(data_size);
    thrust::copy(h_vec_key_1.begin(), h_vec_key_1.end(), d_vec_key_1.begin());
    thrust::copy(h_vec_value_1.begin(), h_vec_value_1.end(), d_vec_value_1.begin());

    thrust::device_vector<unsigned long long> d_vec_key_2(data_size);
    thrust::device_vector<float> d_vec_value_2(data_size);
    thrust::copy(h_vec_key_2.begin(), h_vec_key_2.end(), d_vec_key_2.begin());
    thrust::copy(h_vec_value_2.begin(), h_vec_value_2.end(), d_vec_value_2.begin());
    
    /* reduction 1 */
    thrust::sort_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin());
    thrust::device_vector<unsigned long long> d_vec_key_out_1(data_size);
    thrust::device_vector<float> d_vec_value_out_1(data_size);

    auto new_end_1 = thrust::reduce_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin(),
       	       	 		       d_vec_key_out_1.begin(), d_vec_key_out_1.begin());

    int new_size_r_1 = new_end_1.first - d_vec_key_out_1.begin();

    /* reduction 2 */
    thrust::sort_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin());
    thrust::device_vector<unsigned long long> d_vec_key_out_2(data_size);
    thrust::device_vector<float> d_vec_value_out_2(data_size);

    auto new_end_2 = thrust::reduce_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin(),
       	       	 		       d_vec_key_out_2.begin(), d_vec_key_out_2.begin());

    int new_size_r_2 = new_end_2.first - d_vec_key_out_2.begin();

    int k = 10;
    int number_of_iterations = 1000;
    // int counter = 0;

    thrust::device_vector<float> d_x(new_size_r_2);
    thrust::device_vector<float> d_y(new_size_r_2);
    
    thrust::device_vector<int> d_clusterNo(new_size_r_2);

    thrust::copy(d_vec_value_out_1.begin(), d_vec_value_out_1.end(), d_x.begin());
    thrust::copy(d_vec_value_out_2.begin(), d_vec_value_out_2.end(), d_y.begin());

    thrust::host_vector<float> h_x(new_size_r_2);
    thrust::host_vector<float> h_y(new_size_r_2);
    std::mt19937 rng(std::random_device{}());
    std::shuffle(h_x.begin(), h_x.end(), rng);
    std::shuffle(h_y.begin(), h_y.end(), rng);
    thrust::device_vector<float> d_mean_x(h_x.begin(), h_x.begin() + k);
    thrust::device_vector<float> d_mean_y(h_y.begin(), h_y.begin() + k);

    thrust::device_vector<float> d_sums_x(k);
    thrust::device_vector<float> d_sums_y(k);
    thrust::device_vector<int> d_counts(k, 0);

    const size_t number_of_elements = new_size_r_2;
    const int threads = 1024;
    const int blocks = (number_of_elements + threads - 1) / threads;

    const auto start = std::chrono::high_resolution_clock::now();
    for (size_t iteration = 0; iteration < number_of_iterations; ++iteration) {
     thrust::fill(d_sums_x.begin(), d_sums_x.end(), 0);
     thrust::fill(d_sums_y.begin(), d_sums_y.end(), 0);
     thrust::fill(d_counts.begin(), d_counts.end(), 0);

     assign_clusters<<<blocks, threads>>>(d_x.data(),
                                         d_y.data(),
                                         number_of_elements,
                                         d_mean_x.data(),
                                         d_mean_y.data(),
                                         d_sums_x.data(),
                                         d_sums_y.data(),
                                         k,
                                         d_counts.data(),
					 d_clusterNo.data());

     hipDeviceSynchronize();

     compute_new_means<<<1, k>>>(d_mean_x.data(),
                                d_mean_y.data(),
                                d_sums_x.data(),
                                d_sums_y.data(),
                                d_counts.data());
    hipDeviceSynchronize();
    }

  const auto end = std::chrono::high_resolution_clock::now();
  const auto duration =
  std::chrono::duration_cast<std::chrono::duration<float>>(end - start);
  // std::cerr << "Took: " << duration.count() << "s" << std::endl;

  thrust::host_vector<float> h_mean_x = d_mean_x;
  thrust::host_vector<float> h_mean_y = d_mean_y;
  thrust::host_vector<int> h_counts = d_counts;
  thrust::host_vector<int> h_clusterNo(d_clusterNo.size());

  float distance;
  int best_cluster;

  for(int i = 0; i < new_size_r_2; i++)
  {
	float best_distance = FLT_MAX;
	for (int cluster = 0; cluster < k; ++cluster) {	
    	    distance = squared_l2_distance_h(h_x[i], h_y[i], h_mean_x[cluster], h_mean_y[cluster]);
	    // std::cout << h_x[i] << "," << h_y[i] << "," << cluster << "," << distance << endl;
	    
	    if (distance < best_distance) {
      	      	 best_distance = distance;
      		 best_cluster = cluster;
             }
 
	}	
	// std::cout << "*" << h_x[i] << "," << h_y[i] << "," << best_cluster << "," << distance << endl;
	h_clusterNo[i] = best_cluster;
  }

  std::string fname_clstr = "clustered_" + thread_id;
  // std::remove(fname_clstr);
  ofstream outputfile(fname_clstr);  

  int sum;

  int nBytes = k * sizeof(float);

  float *percent;
  percent = (float *)malloc(nBytes);
  
  for (size_t cluster = 0; cluster < k; ++cluster) {
    sum = sum + h_counts[cluster];
  }

  for (size_t cluster = 0; cluster < k; ++cluster) {
    percent[cluster] = (float)h_counts[cluster] / (float)sum;
  }

  thrust::host_vector<unsigned long long> h_vec_key_f(new_size_r_2);
  thrust::copy(d_vec_key_out_2.begin(),d_vec_key_out_2.end(),h_vec_key_f.begin());


  for(int i=0; i < new_size_r_2; i++)
  {
	outputfile << h_vec_key_f[i] << "," << h_x[i] << "," << h_y[i] << ", cluster" << h_clusterNo[i] << ",(" << percent[h_clusterNo[i]] << "%)" << std::endl;
  }

  outputfile.close();

    /*
    thrust::host_vector<unsigned long long> h_vec_key_2(data_size);
    thrust::host_vector<long> h_vec_value_2(data_size);

    thrust::copy(d_vec_value_out.begin(),d_vec_value_out.end(),h_vec_value_2.begin());
    thrust::copy(d_vec_key_out.begin(),d_vec_key_out.end(),h_vec_key_2.begin());

    for(int i = 0; i < new_size_r; i++)
    {
    	key_out[i] =  h_vec_key_2[i];
	value_out[i] =  h_vec_value_2[i];
    }

    clock_t end_t = clock();
    const double time = static_cast<double>(end_t - start_t) / CLOCKS_PER_SEC * 1000.0;
    cout << "thread:" << thread_id << " - reduction done with new_size " << new_size_r
    	 << "(" << data_size << ") - " << time << endl;

    (*new_size) = new_size_r;
    */
}

void kernel(long *h_key, long *h_value_1, long *h_value_2, string filename, int size)
{
  int N = size;

  cout << "kernel" << endl;

  for(int i = 0; i < 5; i++)
  {
	cout << h_key[i] << "," << h_value_1[i] << endl;
  }

  thrust::host_vector<int> h_vec_1(N);
  std::generate(h_vec_1.begin(), h_vec_1.end(), rand); 

  thrust::device_vector<int> key_in(N);
  thrust::copy(h_vec_1.begin(), h_vec_1.end(), key_in.begin()); 

  thrust::host_vector<unsigned long long> h_vec_key_1(N);
  thrust::host_vector<unsigned long long> h_vec_key_2(N);

  thrust::host_vector<long> h_vec_value_1(N);
  thrust::host_vector<long> h_vec_value_2(N);

  cout << N << endl;

  for(int i=0; i < N; i++)
  {
	// cout << h_key[i] << endl;
	h_vec_key_1[i] = h_key[i];
	h_vec_key_2[i] = h_key[i];
	h_vec_value_1[i] = h_value_1[i];
	h_vec_value_2[i] = h_value_2[i];
  }

  /* 1 -> 3 */

  thrust::device_vector<unsigned long long> d_vec_key_1(N);
  thrust::device_vector<long> d_vec_value_1(N);
  thrust::copy(h_vec_key_1.begin(), h_vec_key_1.end(), d_vec_key_1.begin());
  thrust::copy(h_vec_value_1.begin(), h_vec_value_1.end(), d_vec_value_1.begin());

  // thrust::sort_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin(), thrust::greater<unsigned long long>());

  thrust::sort_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin());

  thrust::host_vector<unsigned long long> h_vec_key_3(N);
  thrust::host_vector<long> h_vec_value_3(N);

  thrust::copy(d_vec_value_1.begin(),d_vec_value_1.end(),h_vec_value_3.begin());
  thrust::copy(d_vec_key_1.begin(),d_vec_key_1.end(),h_vec_key_3.begin());

  /* 2 -> 4 */

  thrust::device_vector<unsigned long long> d_vec_key_2(N);
  thrust::device_vector<long> d_vec_value_2(N);
  thrust::copy(h_vec_key_2.begin(), h_vec_key_2.end(), d_vec_key_2.begin());
  thrust::copy(h_vec_value_2.begin(), h_vec_value_2.end(), d_vec_value_2.begin());

  // thrust::sort_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin(), thrust::greater<unsigned long long>());

  thrust::sort_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin());

  thrust::host_vector<unsigned long long> h_vec_key_4(N);
  thrust::host_vector<long> h_vec_value_4(N);

  thrust::copy(d_vec_value_2.begin(),d_vec_value_2.end(),h_vec_value_4.begin());
  thrust::copy(d_vec_key_2.begin(),d_vec_key_2.end(),h_vec_key_4.begin());

  cout << "1 -> 3" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_3[i] << "," << h_vec_value_3[i] << endl;
  }

  cout << "2 -> 4" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_4[i] << "," << h_vec_value_4[i] << endl;
  }

  thrust::device_vector<unsigned long long> d_vec_key_1_out(N);
  thrust::device_vector<long> d_vec_value_1_out(N);

  thrust::device_vector<unsigned long long> d_vec_key_2_out(N);
  thrust::device_vector<long> d_vec_value_2_out(N);

  auto new_end_1 = thrust::reduce_by_key(d_vec_key_1.begin(), d_vec_key_1.end(), d_vec_value_1.begin(),
       	       	 		       d_vec_key_1_out.begin(), d_vec_value_1_out.begin());

  int new_size_1 = new_end_1.first - d_vec_key_1_out.begin() + 1; 

  thrust::host_vector<unsigned long long> h_vec_key_3_out(N);
  thrust::host_vector<long> h_vec_value_3_out(N);

  thrust::copy(d_vec_value_1_out.begin(),d_vec_value_1_out.end(),h_vec_value_3_out.begin());
  thrust::copy(d_vec_key_1_out.begin(),d_vec_key_1_out.end(),h_vec_key_3_out.begin());

  auto new_end_2 = thrust::reduce_by_key(d_vec_key_2.begin(), d_vec_key_2.end(), d_vec_value_2.begin(),
       	       	 		       d_vec_key_2_out.begin(), d_vec_value_2_out.begin());      

  int new_size_2 = new_end_2.first - d_vec_key_2_out.begin();// + 1; 

  thrust::host_vector<unsigned long long> h_vec_key_4_out(N);
  thrust::host_vector<long> h_vec_value_4_out(N);

  thrust::copy(d_vec_value_2_out.begin(),d_vec_value_2_out.end(),h_vec_value_4_out.begin());
  thrust::copy(d_vec_key_2_out.begin(),d_vec_key_2_out.end(),h_vec_key_4_out.begin());

  cout << "1 -> 3" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_3_out[i] << "," << h_vec_value_3_out[i] << endl;
  }

  cout << "2 -> 4" << endl;
  for(int i = 0; i < 5; i++)
  {
	cout << h_vec_key_4_out[i] << "," << h_vec_value_4_out[i] << endl;
  }

  ofstream outputfile(filename);
    
  cout << "all" << endl;

  outputfile << "timestamp, counted, bytes" << endl;
  
  for(int i = 0; i < new_size_2; i++)
  {
	// cout << h_vec_key_3_out[i] << "," << h_vec_value_3_out[i] << "," << h_vec_value_4_out[i] << endl;

	/*
	if(h_vec_key_3_out[i] != 0)
		outputfile << h_vec_key_3_out[i] << "," << h_vec_value_3_out[i] << "," << h_vec_value_4_out[i] << endl;
	*/
	
	std::string timestamp = to_string(h_vec_key_3_out[i]);

	outputfile << timestamp.substr(0,4) << "-" << timestamp.substr(4,2) << "-" << timestamp.substr(6,2) << " "
	     	   << timestamp.substr(8,2) << ":" << timestamp.substr(10,2) << ":" << timestamp.substr(12,2)
	     	   << "." << timestamp.substr(14,3) << "," 
		   << h_vec_value_3_out[i] << "," << h_vec_value_4_out[i] << endl;
  }

  outputfile.close();
  
}


